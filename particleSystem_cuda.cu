#include "hip/hip_runtime.h"


// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"


#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <helper_functions.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel_impl.cuh"

extern "C"
{

    void cudaInit(int argc, char **argv)
    {
        int devID;

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        devID = findCudaDevice(argc, (const char **)argv);

        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }

    void allocateArray(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void threadSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }

    void copyArrayToDevice(void *device, void *host, int size)
    {
        checkCudaErrors(hipMemcpy(device, host, size, hipMemcpyHostToDevice));
    }

    void copyArrayFromDevice(void* device, void* host, int size)
    {
        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    }

    void setParameters(SimParams *hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

    void calcHash(uint  *gridParticleHash,
                  uint  *gridParticleIndex,
                  float *pos,
                  int    numParticles)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
                                               gridParticleIndex,
                                               (float3 *) pos,
                                               numParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    void reorderDataAndFindCellStart(uint  *cellStart,
                                     uint  *cellEnd,
                                     float *sortedPos,
                                     float *sortedVel,
                                     uint  *gridParticleHash,
                                     uint  *gridParticleIndex,
                                     float *oldPos,
                                     float *oldVel,
                                     uint   numParticles,
                                     uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

        uint smemSize = sizeof(uint)*(numThreads+1);
        reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
            cellStart,
            cellEnd,
            (float3 *) sortedPos,
            (float4 *) sortedVel,
            gridParticleHash,
            gridParticleIndex,
            (float3 *) oldPos,
            (float4 *) oldVel,
            numParticles);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

    }

    void collide(float *newVel,
                 float *newPos,
                 float *sortedVel,
                 float *sortedPos,
                 uint  *cellStart,
                 uint  *cellEnd,
                 uint   numParticles_new,
                 uint   numCells)
    {

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles_new, 64, numBlocks, numThreads);

        // execute the kernel
        collideD<<< numBlocks, numThreads >>>((float3 *)newVel,
                                              (float3 *)newPos,
                                              (float4 *)sortedVel,
                                              (float3*)sortedPos,
                                              cellStart,
                                              cellEnd,
                                              numParticles_new);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

    }


    void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                            thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                            thrust::device_ptr<uint>(dGridParticleIndex));
    }

}   // extern "C"
